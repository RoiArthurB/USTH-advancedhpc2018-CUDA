#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {

}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    
}

void Labwork::labwork3_GPU() {
   
}

void Labwork::labwork4_GPU() {
   
}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

void Labwork::labwork5_GPU() {
    
}

void Labwork::labwork6_GPU() {

}

__global__ void grayscale2D(uchar3 *input, int *histo, int imgWidth, int imgHeight) {
    //Calculate tid
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx >= imgWidth || tidy >= imgHeight) return;

    int localtid =  tidx + (tidy * imgWidth);

    //Process pixel
    unsigned int g = ((int)input[localtid].x + (int)input[localtid].y + (int)input[localtid].z) / 3;
    histo[localtid] = g;
}
__global__ void stretching(int *input, uchar3 *output, int imgWidth, int imgHeight, int min, int max) {
    //Calculate tid
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx >= imgWidth || tidy >= imgHeight) return;

    int localtid =  tidx + (tidy * imgWidth);
    

    //Process pixel
    float num = (input[localtid] - min);
    float gStretch = ((num / (max - min)) * 255);
    
    //Store to output image
    output[localtid].z = output[localtid].y = output[localtid].x = (char)gStretch;
}
void Labwork::labwork7_GPU() {
    // GRAYSCALING
    //======================
    
    // Preparing var
    //----------------------
    //Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
//    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    uchar3 *devInput;
    int *devHisto;

    //Allocate CUDA memory    
    hipMalloc(&devInput, pixelCount * sizeof(uchar3));
    hipMalloc(&devHisto, pixelCount * sizeof(int));
    // Copy CUDA Memory from CPU to GPU
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
    
    //Create 32x32 Blocks
    dim3 blockSize = dim3(32, 32);
    dim3 gridSize = dim3((inputImage->width + (blockSize.x-1))/blockSize.x, 
        (inputImage->height  + (blockSize.y-1))/blockSize.y);

    // Processing
    //----------------------
    // Start GPU processing (KERNEL)
    grayscale2D<<<gridSize, blockSize>>>(devInput, devHisto, inputImage->width, inputImage->height);

    // Cleaning
    //----------------------
    hipFree(&devInput);

    //======================
    // !GRAYSCALING

    // REDUCE
    //======================

    // Prep
    //----------------------
    int *temp = static_cast<int *>(malloc(pixelCount * sizeof(int)));    
    hipMemcpy(temp, devHisto, pixelCount * sizeof(int), hipMemcpyDeviceToHost);
    int hostMax = 0;
    int hostMin = 255;

    // Processing
    //----------------------    
    for (int i = 0; i < pixelCount; i++){
        hostMax = max(hostMax, temp[i]);
        hostMin = min(hostMin, temp[i]);
    }

    // Cleaning
    //----------------------
    // Free CPU Memory
    free(temp);

    //======================
    // !REDUCE


    // STRETCHING
    //======================
    // Prep
    //----------------------
    //Calculate number of pixels
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    uchar3 *devGray;

    //Allocate CUDA memory    
    hipMalloc(&devGray, pixelCount * sizeof(uchar3));

    // Processing
    //----------------------
    // Start GPU processing (KERNEL)
    stretching<<<gridSize, blockSize>>>(devHisto, devGray, inputImage->width, inputImage->height, hostMin, hostMax);
    
    // Copy CUDA Memory from GPU to CPU
    hipMemcpy(outputImage, devGray, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);
    
    // Cleaning
    //----------------------
    // Free CUDA Memory
    hipFree(&devHisto);
    hipFree(&devGray);
    //======================
    // !STRETCHING
}

typedef struct hsv {
    double *h, *s, *v;
} Hsv ;
__global__ void RGB2HSV(uchar3 *in, Hsv out, int imgWidth, int imgHeight) {
    //Calculate tid
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx >= imgWidth || tidy >= imgHeight) return;
    
    int tid =  tidx + (tidy * imgWidth);
    double s, v, h = 0;
    
    // Scaling from [0 .. 255] to [0 .. 1]
    // Local var for optimization
    double pixelR = (double)in[tid].x / 255.0;
    double pixelG = (double)in[tid].y / 255.0;
    double pixelB = (double)in[tid].z / 255.0;
    
    double pxMax = max(pixelR, max(pixelG, pixelB));
    //int pxMin = // No need of this variable => Only 1 use 
    
    // V
    //======================

    v = pxMax;
    
    //======================
    // ! V

    // S
    //======================
    double delta = pxMax - min(pixelR, min(pixelG, pixelB));

    if( pxMax <= 0.0 ) { // NOTE: if Max is == 0, this divide would cause a crash
        // if max is 0, then r = g = b = 0              
        // s = 0, h is undefined
        s = 0.0;
    } else {
        s = (delta / pxMax);
    }
    //======================
    // ! S

    // H
    //======================
    if( pixelR >= pxMax ){ // between yellow & magenta
        h = ( pixelG - pixelB ) / delta;
        int decimal = static_cast<int>(h*10)%10;
        h = ((int)h % 6) + decimal * 0.1;
    }else{
        if( pixelG >= pxMax )
            h = 2.0 + ( ( pixelB - pixelR ) / delta );  // between cyan & yellow
        else
            h = 4.0 + ( ( pixelR - pixelG ) / delta );  // between magenta & cyan
    }
    
    // degrees
    h *= 60.0;
    //======================
    // ! H

    // Save new val in SoA
    out.h[tid] = h;
    out.s[tid] = s;
    out.v[tid] = v;
}
__global__ void HSV2RGB(Hsv in, uchar3 *out, int imgWidth, int imgHeight) {
    //Calculate tid
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx >= imgWidth || tidy >= imgHeight) return;
    
    int tid =  tidx + (tidy * imgWidth);
    
    // Prepare local value for optimization
    double pixelH = in.h[tid];
    double pixelS = in.s[tid];
    double pixelV = in.v[tid];
    
    double d = pixelH / 60.0;
    // No need hi => Only 1 use
    double f = d - ((int)d % 6);
    
    double l = pixelV * (1.0 - pixelS);
    double m = pixelV * (1.0 - f * pixelS);
    double n = pixelV * (1.0 - (1.0 - f) * pixelS);
    
    // Calculate RGB values
    double r, g, b;
    switch ((int)pixelH / 60){
        case 0:
            r = pixelV;
            g = n;
            b = l;
            break;
        case 1:
            r = m;
            g = pixelV;
            b = l;
            break;
        case 2:
            r = l;
            g = pixelV;
            b = n;
            break;
        case 3:
            r = l;
            g = m;
            b = pixelV;
            break;
        case 4:
            r = n;
            g = l;
            b = pixelV;
            break;
        case 5:
        default:
            r = pixelV;
            g = l;
            b = m;
            break;
    }
    
    //Note : out[].x = R | out[].y = G | out[].z = B
    // [0..1] to [0..255]
    out[tid].x = (char)(r * 255);
    out[tid].y = (char)(g * 255);
    out[tid].z = (char)(b * 255);
}
void Labwork::labwork8_GPU() {

    // GRAYSCALING
    //======================
    
    // Preparing var
    //----------------------
    //Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
    //Kernel param
    dim3 blockSize = dim3(32, 32);
    dim3 gridSize = dim3((inputImage->width + (blockSize.x-1))/blockSize.x, 
        (inputImage->height  + (blockSize.y-1))/blockSize.y);
    
    //Kernel var
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    Hsv hsvArray;
    
    // Malloc arrays inside the structure
    hipMalloc((void**)&hsvArray.h, pixelCount * sizeof(double));
    hipMalloc((void**)&hsvArray.s, pixelCount * sizeof(double));
    hipMalloc((void**)&hsvArray.v, pixelCount * sizeof(double));
    
    uchar3 *devInput; 
    hipMalloc(&devInput, pixelCount * sizeof(uchar3));
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    // Processing
    //----------------------
    // Start GPU processing (KERNEL)
    RGB2HSV<<<gridSize, blockSize>>>(devInput, hsvArray, inputImage->width, inputImage->height);
    // No need to clean devInput
    HSV2RGB<<<gridSize, blockSize>>>(hsvArray, devInput, inputImage->width, inputImage->height);
    
    // Get final image
    hipMemcpy(outputImage, devInput, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);

    // Cleaning
    //----------------------
    hipFree(devInput);
    hipFree(hsvArray.h);
    hipFree(hsvArray.s);
    hipFree(hsvArray.v);

    //======================
    // !GRAYSCALING
    
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
